#include <chrono>   // for timing
#include <iostream> // std::cout, std::endl
#include <iterator> // std::ostream_iterator
#include <vector>
#include "hip/hip_runtime.h"
#include ""

#include "dense_graph.h"
#include "sparse_graph.h"

#include "data_generator.h"
#include "data_types.h"

/**
 * Runs timing tests on a CUDA graph implementation.
 * Consists of independently constructing the graph and then
 * modifying it to its two-hop neighbourhood.
 */
template < typename DeviceGraph >
void run(DeviceGraph g, csc485b::a2::edge_t const* d_edges, std::size_t m)
{

    int const warp_size = 32;
    int const products_per_block = 1024 / warp_size;
    int const num_blocks = g.n * m / products_per_block;

    hipDeviceSynchronize();
    auto const build_start = std::chrono::high_resolution_clock::now();

    // this code doesn't work yet!
    /*csc485b::a2::gpu::build_graph << < num_blocks, threads_per_block, threads_per_block * sizeof(csc485b::a2::edge_t) >> > (g, d_edges, m);*/
    csc485b::a2::gpu::build_graph << < num_blocks, dim3{warp_size, products_per_block} >> > (g, d_edges, m);

    hipDeviceSynchronize();
    auto const reachability_start = std::chrono::high_resolution_clock::now();

    // neither does this!
    csc485b::a2::gpu::two_hop_reachability << < num_blocks, dim3{ warp_size, products_per_block } >> > (g);

    hipDeviceSynchronize();
    auto const end = std::chrono::high_resolution_clock::now();

    std::cout << "Build time: "
        << std::chrono::duration_cast<std::chrono::microseconds>(reachability_start - build_start).count()
        << " us"
        << std::endl;

    std::cout << "Reachability time: "
        << std::chrono::duration_cast<std::chrono::microseconds>(end - reachability_start).count()
        << " us"
        << std::endl;
}

/**
 * Allocates space for a dense graph and then runs the test code on it.
 */
void run_dense(csc485b::a2::edge_t const* d_edges, std::size_t n, std::size_t m)
{
    using namespace csc485b;

    // allocate device DenseGraph
    a2::node_t* d_matrix;
    hipMalloc((void**)&d_matrix, sizeof(a2::node_t) * n * n);
    a2::DenseGraph d_dg{ n, d_matrix };

    run(d_dg, d_edges, m);

    // check output?
    std::vector< a2::node_t > host_matrix(d_dg.matrix_size());
    a2::DenseGraph dg{ n, host_matrix.data() };
    hipMemcpy(dg.adjacencyMatrix, d_dg.adjacencyMatrix, sizeof(a2::node_t) * d_dg.matrix_size(), hipMemcpyDeviceToHost);
    std::copy(host_matrix.cbegin(), host_matrix.cend(), std::ostream_iterator< a2::node_t >(std::cout, " "));

    // clean up
    hipFree(d_matrix);
}

/**
 * Allocates space for a sparse graph and then runs the test code on it.
 */
void run_sparse(csc485b::a2::edge_t const* d_edges, std::size_t n, std::size_t m)
{
    using namespace csc485b;

    // allocate device SparseGraph
    a2::node_t* d_offsets, * d_neighbours;
    hipMalloc((void**)&d_offsets, sizeof(a2::node_t) * n);
    hipMalloc((void**)&d_neighbours, sizeof(a2::node_t) * m);
    a2::SparseGraph d_sg{ n, m, d_offsets, d_neighbours };

    run(d_sg, d_edges, m);

    // clean up
    hipFree(d_neighbours);
    hipFree(d_offsets);
}

int main()
{
    using namespace csc485b;

    // Create input
    std::size_t constexpr n = 4;
    std::size_t constexpr expected_degree = n >> 1;

    a2::edge_list_t const graph = a2::generate_graph(n, n * expected_degree);
    std::size_t const m = graph.size();

    // lazily echo out input graph
    for (auto const& e : graph)
    {
        std::cout << "(" << e.x << "," << e.y << ") ";
    }

    // allocate and memcpy input to device
    a2::edge_t* d_edges;
    hipMalloc((void**)&d_edges, sizeof(a2::edge_t) * m);
    hipMemcpyAsync(d_edges, graph.data(), sizeof(a2::edge_t) * m, hipMemcpyHostToDevice);

    // run your code!
    run_dense(d_edges, n, m);
    run_sparse(d_edges, n, m);

    return EXIT_SUCCESS;
}